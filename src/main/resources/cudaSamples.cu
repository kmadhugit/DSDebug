
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
        printf("CUDA KERNEL ADD %ld + %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void mul(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
        printf("CUDA KERNEL MUL %ld * %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void square(int n, long *a, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * a[i];
        printf("CUDA KERNEL square %ld * %ld = %ld \n",a[i],a[i],sum[i]);
    }

}
